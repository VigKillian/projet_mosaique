#include "hip/hip_runtime.h"
#include <stdio.h>
#include "image_ppm.h"
#include <iostream>
#include <string>
#include <algorithm>
#include <map>
#include <cmath>
#include <float.h>
#include <hip/hip_runtime.h>
#define NB_BASE_DE_DONNEE 10000
using namespace std;
typedef unsigned char OCTET;

__global__ void computeHistoKernel(
    OCTET* d_imgIn,
    int* d_histos,
    int nW,
    int nH,
    int tailleBloc,
    int nbImagettes
){


}

__global__ void matchImagetteKernel(
  OCTET* d_imgIn,
  float* d_moyennesImagettes,
  int* d_resultImagetteIds,
  int nW,
  int nH,
  int tailleBloc,
  int nbImagettes
){
  int tileX = blockIdx.x*blockDim.x+threadIdx.x;
  int tileY = blockIdx.y*blockDim.y+threadIdx.y;

  int nbTileX = nW/tailleBloc;
  int nbTileY = nH/tailleBloc;

  if (tileX >= nbTileX || tileY >= nbTileY) return;

  int startX = tileX * tailleBloc;
  int startY = tileY * tailleBloc;

  float sum = 0.f;
  for(int dy = 0;dy<tailleBloc;dy++){
    for(int dx = 0;dx<tailleBloc;dx++){
      int px = startX + dx;
      int py = startY + dy;
      sum += d_imgIn[py*nW+px];
    }
  }
  sum /= (tailleBloc*tailleBloc);
  float minDiff = FLT_MAX;
  int bestId = -1;
  for (int i = 0; i < nbImagettes; i++) {
    float diff = fabsf(d_moyennesImagettes[i] - sum);
    if (diff < minDiff) {
      minDiff = diff;
      bestId = i;
    }
  }

  d_resultImagetteIds[tileY * nbTileX + tileX] = bestId;

}

int main(int argc, char* argv[])
{
  char cNomImgLue[250], cNomImgEcrite[250];
  int nH, nW, nTaille, tailleBloc;
  float moyennesImagettes[NB_BASE_DE_DONNEE];

  if (argc != 4) 
    {
      printf("Usage: ImageIn.pgm ImageOut.pgm taille_bloc \n"); 
      exit (1);
    }

  std::chrono::time_point<std::chrono::high_resolution_clock> _t0 = std::chrono::high_resolution_clock::now();
  
  sscanf (argv[1],"%s",cNomImgLue);
  sscanf (argv[2],"%s",cNomImgEcrite);
  sscanf (argv[3],"%d",&tailleBloc);

  OCTET *ImgIn, *ImgOut;

  lire_nb_lignes_colonnes_image_pgm(cNomImgLue, &nH, &nW);
  nTaille = nH * nW;

  allocation_tableau(ImgIn, OCTET, nTaille);
  lire_image_pgm(cNomImgLue, ImgIn, nH * nW);
  allocation_tableau(ImgOut, OCTET, nTaille);

  // Charger les imagettes et calculer leur moyenne de luminosité
  for(int idImagette = 0; idImagette < NB_BASE_DE_DONNEE; idImagette++){
    OCTET *ImgIn_imagette;
    int nH_imagette, nW_imagette, nTaille_imagette;
    
    loadImagette(idImagette+1, ImgIn_imagette, nH_imagette, nW_imagette, nTaille_imagette);
    
    float moyenneImagette = 0.f;
    for(int j = 0; j < nTaille_imagette; j++){
      moyenneImagette += ImgIn_imagette[j];
    }
    moyenneImagette /= float(nTaille_imagette);
    moyennesImagettes[idImagette] = moyenneImagette;
    
    free(ImgIn_imagette);
  }


  // Remplacement des blocs de l'image originale
  OCTET* d_imgIn;
  float* d_moyennesImagettes;
  int* d_resultImagetteIds;
  int nbTile = nH*nW/tailleBloc/tailleBloc;
  hipMalloc(&d_imgIn,sizeof(OCTET)*nTaille);
  hipMalloc(&d_moyennesImagettes,sizeof(float)*NB_BASE_DE_DONNEE);
  hipMalloc(&d_resultImagetteIds,sizeof(int)*nbTile);
  hipMemcpy(d_imgIn, ImgIn, sizeof(OCTET) * nTaille, hipMemcpyHostToDevice);
  hipMemcpy(d_moyennesImagettes,moyennesImagettes,sizeof(float)*NB_BASE_DE_DONNEE,hipMemcpyHostToDevice);


  dim3 threadsPerBlock(16,16);
  dim3 nbBlock((nW/tailleBloc+15)/16,(nH/tailleBloc+15)/16);
  matchImagetteKernel<<<nbBlock,threadsPerBlock>>>(
    d_imgIn,d_moyennesImagettes,d_resultImagetteIds,nW,nH,tailleBloc,NB_BASE_DE_DONNEE
  );
  hipDeviceSynchronize(); //attend que toutes les operations soient terminees.
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
  }
  // copy back to host
  int* resultImagetteIds = new int[nbTile];
  hipMemcpy(resultImagetteIds,d_resultImagetteIds,sizeof(int)*nbTile,hipMemcpyDeviceToHost);

  for(int i = 0; i <= nH - tailleBloc; i += tailleBloc){
    for(int j = 0; j <= nW - tailleBloc; j += tailleBloc){
      int pixelDepart = i * nW + j;
      OCTET *ImgOut_imagette;
      OCTET *ImgIn_imagette;
      int nH_imagette, nW_imagette, nTaille_imagette;
      
      loadImagette(resultImagetteIds[i/tailleBloc*nW/tailleBloc+j/tailleBloc]+1, ImgIn_imagette, nH_imagette, nW_imagette, nTaille_imagette);
      allocation_tableau(ImgOut_imagette, OCTET, tailleBloc * tailleBloc);
      resize_imagette(ImgIn_imagette, nH_imagette, nW_imagette, ImgOut_imagette, tailleBloc, tailleBloc);
      
      // Remplacer les pixels du bloc par ceux de l'imagette
      for(int k = 0; k < tailleBloc; k++){
        for(int p = 0; p < tailleBloc; p++){
          ImgOut[pixelDepart + k * nW + p] = ImgOut_imagette[k * tailleBloc + p];
        }
      }
      
      free(ImgIn_imagette);
      free(ImgOut_imagette);
    }
  }

  ecrire_image_pgm(cNomImgEcrite, ImgOut, nH, nW);
  

  std::chrono::time_point<std::chrono::high_resolution_clock> _t1 = std::chrono::high_resolution_clock::now();
  std::cout << "Photo-mosaïque produite en : "<<std::chrono::duration<double>(_t1-_t0).count() << "s" << std::endl;

  std::cout<<"PSNR par rapport à l'image d'entrée : "<<calculer_PSNR(ImgIn, ImgOut, nTaille)<<"dB"<<std::endl;
  free(ImgIn);
  free(ImgOut);

  return 1;
}